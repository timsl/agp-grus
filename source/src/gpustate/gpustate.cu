#include "hip/hip_runtime.h"
#include "gpustate.cuh"

void GPUState::init(const CUParticle *in, size_t n, GLuint vbo_sphere_inst) {
  CUDAERR(hipMalloc(&particles, n * sizeof(*particles)));
  CUDAERR(hipMalloc(&velocities, n * sizeof(*velocities)));

  // glm is memory compatible with float3 so a struct should be aswell
  CUDAERR(hipMemcpy(particles, in, n * sizeof(*particles),
                     hipMemcpyHostToDevice));

  CUDAERR(hipGraphicsGLRegisterBuffer(&resources, vbo_sphere_inst,
                                       cudaGraphicsMapFlagsNone));
  CUDAERR(hipGraphicsMapResources(1, &resources));
  size_t size = 0;
  CUDAERR(hipGraphicsResourceGetMappedPointer(&glptr, &size, resources));
}

void GPUState::clean() {
  CUDAERR(hipGraphicsUnmapResources(1, &resources));
  CUDAERR(hipGraphicsUnregisterResource(resources));
  CUDAERR(hipFree(particles));
  CUDAERR(hipFree(velocities));
}