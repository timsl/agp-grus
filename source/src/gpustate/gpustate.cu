#include "hip/hip_runtime.h"
#include "gpustate.cuh"

void GPUState::init(const CUParticle *in, size_t n) {
  CUDAERR(hipMalloc(&particles, n * sizeof(*particles)));
  CUDAERR(hipMalloc(&velocities, n * sizeof(*velocities)));

  // glm is memory compatible with float3 so a struct should be aswell
  CUDAERR(hipMemcpy(particles, in, n * sizeof(*particles),
                     hipMemcpyHostToDevice));
}
void GPUState::clean() {
  CUDAERR(hipFree(particles));
  CUDAERR(hipFree(velocities));
}