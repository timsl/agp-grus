#include "hip/hip_runtime.h"
#include "kernel.cuh"

__device__ float3 body_body_interaction(CUParticle pi, CUParticle pj) {
  const double D = 376.78;
  const double D2 = pow(D, 2);
  const double epsilon = 47.0975;
  const double M[4] = {1.9549 * pow(10, 10), 7.4161 * pow(10, 9),
                       1.9549 * pow(10, 10), 7.4161 * pow(10, 9)};
  const double K[4] = {5.8228 * pow(10, 14), 2.29114 * pow(10, 14),
                       5.8228 * pow(10, 14), 2.29114 * pow(10, 14)};
  const double KRP[4] = {0.02, 0.01, 0.02, 0.01};
  const double SDP[4] = {0.002, 0.001, 0.002, 0.001};
  const double G = 6.67408; // * 10^-20, but removed that from M and here

  // These scales are probably not necessary if we use better numerical techs
  const double weirdscale1 = pow(10, -16);
  const double weirdscale2 = pow(10, -22);

  const auto diff = pj.pos - pi.pos;
  const auto next_diff =
      ((pj.pos + pj.velocity * 0.00001) - (pi.pos + pi.velocity * 0.00001));

  double r = norm3d(diff.x, diff.y, diff.z);
  const double next_r = norm3d(next_diff.x, next_diff.y, next_diff.z);

  const auto dir = diff / r;
  double force = 0.0;

  // pre-computed values
  r = fmax(r, epsilon);
  const double r2 = pow(r, 2);
  const double gmm = G * M[pi.type] * M[pj.type] * pow(r, -2) * weirdscale1;
  const double dmr = (D2 - r2) * 0.5 * weirdscale2;
  const double oneshell = fmin(SDP[pi.type], SDP[pj.type]);
  const double twoshell = fmax(SDP[pi.type], SDP[pj.type]);

  if (r >= D) {
    // Not in contact
    force = gmm;
  } else if (r >= D - D * oneshell) {
    // In contact, but no shell penetrated
    force = gmm - dmr * (K[pi.type] + K[pj.type]);
  } else if (r >= D - D * twoshell) {
    // One shell has been penetrated
    if (next_r < r) {
      force = gmm - dmr * (K[pi.type] + K[pj.type]);
    } else {
      force = gmm - dmr * (K[pi.type] * KRP[pi.type] + K[pj.type]);
    }
  } else {
    // Both shells penetrated (r > epsilon)
    if (next_r < r) {
      force = gmm - dmr * (K[pi.type] + K[pj.type]);
    } else {
      force =
          gmm - dmr * (K[pi.type] * KRP[pi.type] + K[pj.type] * KRP[pj.type]);
    }
  }

  return dir * (float)force;
}

__global__ void calculate_velocities(const CUParticle *particles,
                                     float3 *velocities, size_t n, float dt) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= n)
    return;

  // Want a local velocity to write to in each loop to save some global
  // bandwidth
  const CUParticle my_part = particles[i];
  float3 vel_acc = make_float3(0.0, 0.0, 0.0);

  extern __shared__ CUParticle sh_part[];

  const size_t sync_size = blockDim.x;
  const size_t sync_points = n / sync_size;

  for (size_t sync = 0; sync < sync_points; ++sync) {
    // read global memory and put in sh_part instead.
    // put in some j corresponding to this threads idx.
    sh_part[threadIdx.x] = particles[sync * sync_size + threadIdx.x];
    __syncthreads();

    for (size_t j = 0; j < sync_size; ++j) {
      if (sync * sync_size + j == i) {
        continue;
      }

      vel_acc += body_body_interaction(my_part, sh_part[j]);
    }
    __syncthreads();
  }

  velocities[i] = vel_acc * dt;
}

__global__ void apply_velocities(CUParticle *particles, float3 *velocities,
                                 size_t n, float dt) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= n)
    return;

  particles[i].velocity += velocities[i];
  particles[i].pos += particles[i].velocity * dt;
}

void update(WorldState *world, float dt) {
  const auto N = world->particles.size();
  const auto block_size = 32;

  calculate_velocities<<<(N + block_size - 1) / block_size, block_size,
                         block_size * sizeof(CUParticle)>>>(
      world->gpu.particles, world->gpu.velocities, N, dt);
  apply_velocities<<<(N + block_size - 1) / block_size, block_size>>>(
      world->gpu.particles, world->gpu.velocities, N, dt);

  update_GL<<<(N + block_size - 1) / block_size, block_size>>>(
      world->gpu.particles, world->gpu.glptr, N);

  // CUDAERR(hipDeviceSynchronize());

  // CUParticle *cast = reinterpret_cast<CUParticle *>(world->particles.data());
  // CUDAERR(hipMemcpy(cast, world->gpu.particles, N * sizeof(*cast),
  //                    hipMemcpyDeviceToHost));
}
