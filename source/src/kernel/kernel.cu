#include "hip/hip_runtime.h"
#include "kernel.cuh"

__device__ float3 body_body_interaction(CUParticle pi, CUParticle pj) {
  const double D = 376.78;
  const double D2 = pow(D, 2);
  const double epsilon = 47.0975;
  const double M[4] = {1.9549 * pow(10, 10), 7.4161 * pow(10, 9),
                       1.9549 * pow(10, 10), 7.4161 * pow(10, 9)};
  const double K[4] = {5.8228 * pow(10, 14), 2.29114 * pow(10, 14),
                       5.8228 * pow(10, 14), 2.29114 * pow(10, 14)};
  const double KRP[4] = {0.02, 0.01, 0.02, 0.01};
  const double SDP[4] = {0.002, 0.001, 0.002, 0.001};
  const double G = 6.67408; // * 10^-20, but removed that from M and here

  // These scales are probably not necessary if we use better numerical techs
  const double weirdscale1 = pow(10, -16);
  const double weirdscale2 = pow(10, -22);

  const float3 p_i = pi.pos;
  const float3 v_i = pi.velocity;
  const char t_i = pi.type;

  const float3 p_j = pj.pos;
  const float3 v_j = pj.velocity;
  const char t_j = pj.type;

  const auto diff = p_j - p_i;
  const auto next_diff = ((p_j + v_j * 0.00001) - (p_i + v_i * 0.00001));

  double r = norm3d(diff.x, diff.y, diff.z);
  const double next_r = norm3d(next_diff.x, next_diff.y, next_diff.z);

  const auto dir = diff / r;
  double force = 0.0;

  // pre-computed values
  r = fmax(r, epsilon);
  const double r2 = pow(r, 2);
  const double gmm = G * M[t_i] * M[t_j] * pow(r, -2) * weirdscale1;
  const double dmr = (D2 - r2) * 0.5 * weirdscale2;
  const double oneshell = fmin(SDP[t_i], SDP[t_j]);
  const double twoshell = fmax(SDP[t_i], SDP[t_j]);

  if (r >= D) {
    // Not in contact
    force = gmm;
  } else if (r >= D - D * oneshell) {
    // In contact, but no shell penetrated
    force = gmm - dmr * (K[t_i] + K[t_j]);
  } else if (r >= D - D * twoshell) {
    // One shell has been penetrated
    if (next_r < r) {
      force = gmm - dmr * (K[t_i] + K[t_j]);
    } else {
      force = gmm - dmr * (K[t_i] * KRP[t_i] + K[t_j]);
    }
  } else {
    // Both shells penetrated (r > epsilon)
    if (next_r < r) {
      force = gmm - dmr * (K[t_i] + K[t_j]);
    } else {
      force = gmm - dmr * (K[t_i] * KRP[t_i] + K[t_j] * KRP[t_j]);
    }
  }

  return dir * (float)force;
}

__global__ void calculate_velocities(const CUParticle *particles,
                                     float3 *velocities, size_t n, float dt) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= n)
    return;

  // Want a local velocity to write to in each loop to save some global
  // bandwidth
  float3 vel_acc = make_float3(0.0, 0.0, 0.0);

  for (size_t j = 0; j < n; ++j) {
    if (j == i) {
      continue;
    }

    vel_acc += body_body_interaction(particles[i], particles[j]);
  }

  velocities[i] = vel_acc * dt;
}

__global__ void apply_velocities(CUParticle *particles, float3 *velocities,
                                 size_t n, float dt) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= n)
    return;

  particles[i].velocity += velocities[i];
  particles[i].pos += particles[i].velocity * dt;
}

void update(WorldState *world, float dt) {
  const auto N = world->particles.size();
  const auto block_size = 256;

  CUParticle *d_p = 0;
  float3 *d_v = 0;

  CUDAERR(hipMalloc(&d_p, N * sizeof(*d_p)));
  CUDAERR(hipMalloc(&d_v, N * sizeof(*d_v)));

  // glm is memory compatible with float3 so a struct should be aswell
  CUParticle *particles =
    reinterpret_cast<CUParticle *>(world->particles.data());
  CUDAERR(hipMemcpy(d_p, particles, N * sizeof(*d_p), hipMemcpyHostToDevice));

  calculate_velocities<<<(N + block_size - 1) / block_size, block_size>>>(
      d_p, d_v, N, dt);
  apply_velocities<<<(N + block_size - 1) / block_size, block_size>>>(d_p, d_v,
                                                                      N, dt);

  CUDAERR(hipMemcpy(particles, d_p, N * sizeof(*particles),
                     hipMemcpyDeviceToHost));

  CUDAERR(hipFree(d_p));
  CUDAERR(hipFree(d_v));
}
